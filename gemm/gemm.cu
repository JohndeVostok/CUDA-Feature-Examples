#include <cstdint>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <chrono>

void gemmCPU(double *A, double *B, double *C, int m, int n, int k, int a, int b) {
    for (int i = 0; i < m; i++) {
        for (int t = 0; t < k; t++) {
            for (int j = 0; j < n; j++) {
                C[i * n + j] += A[i * k + t] * B[t * n + j];
            }
        }
    }
}

void print_matrix(double *A, int m, int n) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%f ", A[i * n + j]);
        }
        printf("\n");
    }
}

int main() {
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    // cublasSetMathMode(handle, CUBLAS_TENSOR_OP_MATH);

//    int m = atoi(getenv("CUOPT_GEMM_M"));
//    int n = atoi(getenv("CUOPT_GEMM_N"));
//    int k = atoi(getenv("CUOPT_GEMM_K"));
    int m = 4096, n = 4096, k = 4096;

    printf("%d %d %d\n", m, n, k);

    uint32_t rowsA = m, colsA = k, rowsB = k, colsB = n, rowsC = m, colsC = n;
    size_t matrixSizeA = (size_t)rowsA * colsA, matrixSizeB = (size_t)rowsB * colsB, matrixSizeC = (size_t)rowsC * colsC;
    double *devPtrA = 0, *devPtrB = 0, *devPtrC = 0, *devPtrD = 0;
    double a = 1, b = 1;
    
    hipMalloc((void**)&devPtrA, matrixSizeA * sizeof(double));
    hipMalloc((void**)&devPtrB, matrixSizeB * sizeof(double));
    hipMalloc((void**)&devPtrC, matrixSizeC * sizeof(double));
    hipMalloc((void**)&devPtrD, matrixSizeC * sizeof(double));
    double *A = (double *)malloc(matrixSizeA * sizeof(double));
    double *B = (double *)malloc(matrixSizeB * sizeof(double));
    double *C = (double *)malloc(matrixSizeC * sizeof(double));
    double *C_cpu = (double *)malloc(matrixSizeC * sizeof(double));
    double *C_gpu = (double *)malloc(matrixSizeC * sizeof(double));
    double *C_tc = (double *)malloc(matrixSizeC * sizeof(double));
    for (int i = 0; i < matrixSizeA; i++) A[i] = rand() % 5;
    for (int i = 0; i < matrixSizeB; i++) B[i] = rand() % 5;
    for (int i = 0; i < matrixSizeC; i++) C[i] = rand() % 5;
    for (int i = 0; i < matrixSizeC; i++) C_cpu[i] = C[i];
    for (int i = 0; i < matrixSizeC; i++) C_gpu[i] = C[i];
    for (int i = 0; i < matrixSizeC; i++) C_tc[i] = C[i];
    
    hipblasSetMatrix(rowsA, colsA, sizeof(double), A, rowsA, devPtrA, rowsA);
    hipblasSetMatrix(rowsB, colsB, sizeof(double), B, rowsB, devPtrB, rowsB);
    hipblasSetMatrix(rowsC, colsC, sizeof(double), C, rowsC, devPtrC, rowsC);
    hipblasSetMatrix(rowsC, colsC, sizeof(double), C, rowsC, devPtrD, rowsC);

    // gemmCPU(A, B, C_cpu, m, n, k, a, b);
    // uint64_t time_tc = 0;
    for (int i = 0; i < 13; i++) {
        hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &a, devPtrA, k, devPtrB, k, &b, devPtrC, m);
    }
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    for (int i = 0; i < 13; i++) {
        hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &a, devPtrA, k, devPtrB, k, &b, devPtrC, m);
    }
    // printf("TC: %f\n", time_tc / 256.0);

    // uint64_t time_gpu = 0;
    // for (int i = 0; i < 1024; i++) {
    //     auto start = std::chrono::system_clock::now();
    //     cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, n, m, k, &a, devPtrB, n, devPtrA, k, &b, devPtrC, n);
    //     auto end = std::chrono::system_clock::now();
    //     time_gpu += (uint64_t)std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
    // }
    // printf("GPU: %lu\n", time_gpu);

    // cublasGetMatrix(rowsC, colsC, sizeof(double), devPtrC, rowsC, C_gpu, rowsC);
    // cublasGetMatrix(rowsC, colsC, sizeof(double), devPtrD, rowsC, C_tc, rowsC);

    // bool flag = true;
    // for (int i = 0; i < m; i++) {
    //     for (int j = 0; j < n; j++) {
    //         if (C_tc[i * n + j] != C_gpu[i * n + j]) {
    //             flag = false;
    //         }
    //     }
    // }
    // if (flag) printf("Validated.\n"); else printf("Unvalidated.\n");

    hipFree(devPtrA);
    hipFree(devPtrB);
    hipFree(devPtrC);
    free(A);
    free(B);
    free(C);

    hipblasDestroy(handle);
    return 0;
}
