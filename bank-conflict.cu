// Shared memory unexpected bank-conflict.

#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <string>

#include <hip/hip_runtime.h>
#include <mma.h>

#include "hipblas.h"

using namespace nvcuda;

__global__ void latency_hiding(float *data, int size, uint64_t *clock) {
    extern __shared__ int4 tmp[];
    uint64_t clock_start = clock64();

    int4 dest;
    dest = tmp[threadIdx.x];
    dest.x += 1;
    dest.y += 1;
    dest.z += 1;
    dest.w += 1;
    tmp[threadIdx.x] = dest;

    uint64_t clock_end = clock64();
    // atomicAdd(reinterpret_cast<unsigned long long *>(clock), clock_end - clock_start);
}

int main(int argc, char *argv[]) {
	hipError_t cuda_status;
	cuda_status = hipSetDevice(0);
	if (cuda_status != hipSuccess) {
		printf("hipSetDevice failed! ");
		return 1;
	}

    half *devPtrA = 0, *devPtrB = 0;
    float *devPtrC = 0, *devPtrD = 0;
    float alpha = 1, beta = 1;
    uint32_t *devPtrDebug = 0;

    hipMalloc((void **) &devPtrA, 65536 * sizeof(half));
    hipMalloc((void **) &devPtrB, 65536 * sizeof(half));
    hipMalloc((void **) &devPtrC, 65536 * sizeof(float));
    hipMalloc((void **) &devPtrD, 65536 * sizeof(float));
    hipMalloc((void **) &devPtrDebug, 65536);
	
    half *ptrA = (half *) malloc(65536 * sizeof(half));
    half *ptrB = (half *) malloc(65536 * sizeof(half));
    float *ptrC = (float *) malloc(65536 * sizeof(float));
    float *ptrD = (float *) malloc(65536 * sizeof(float));
    float *ptrE = (float *) malloc(65536 * sizeof(float));
    uint32_t *ptrDebug = (uint32_t *) malloc(65536);

    memset(ptrC, 0, 65536 * sizeof(float));
    hipMemcpy(devPtrA, ptrA, 65536 * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(devPtrB, ptrB, 65536 * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(devPtrC, ptrC, 65536 * sizeof(float), hipMemcpyHostToDevice);

    uint64_t *clk;

    hipMallocManaged(&clk, sizeof(uint64_t));

    hipError_t result;
    result = hipFuncSetAttribute(reinterpret_cast<const void*>(latency_hiding), hipFuncAttributeMaxDynamicSharedMemorySize, 72000);
    if (result != hipSuccess) {
        return 0;
    }
    result = hipFuncSetAttribute(reinterpret_cast<const void*>(latency_hiding), hipFuncAttributePreferredSharedMemoryCarveout, 100);
    if (result != hipSuccess) {
        return 0;
    }
    latency_hiding<<<160, 1024, 72000>>>(devPtrC, 64, clk);
    hipDeviceSynchronize();
    printf("%lu\n", *clk);

    cuda_status = hipDeviceReset();
	if (cuda_status != hipSuccess) {
		printf("hipDeviceReset failed! ");
		return 1;
	}

	return 0;
}